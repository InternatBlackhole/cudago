
#include <hip/hip_runtime.h>
// bitonično urejanje tabele celih števil
// 		argumenta: število niti v bloku in velikost tabele
//		elementi tabele so inicializirani naključno
// s sinhornizacijo niti v bloku se v največji možni meri izognemo globalni sinhornizaciji
// bitonicSort je zdaj funkcija na napravi, ki jo kličejo trije ščepci
// bitonicSortStart in bitonicSortFinish urejata v skupnem pomnilniku

__device__ void bitonicSort(int *a, int len, int k, int j) {
	int gid = blockIdx.x * blockDim.x + threadIdx.x;    
    if (gid < len/2) {
		int i1 = 2*j * (int)(gid / j) + (gid % j);	// prvi element
		int i2 = i1 ^ j;							// drugi element
		int dec = i1 & k;							// smer urejanja (padajoče: dec != 0)
		if ((dec == 0 && a[i1] > a[i2]) || (dec != 0 && a[i1] < a[i2])) {
			int temp = a[i1];
			a[i1] = a[i2];
			a[i2] = temp;
		}
	}
}

__device__ void bitonicSortShared(int *as, int len, int k, int j) {
	int gid = blockIdx.x * blockDim.x + threadIdx.x;    
    if (gid < len/2) {
		int i1 = 2*j * (int)(gid / j) + (gid % j);	// prvi element
		int i2 = i1 ^ j;							// drugi element
		int dec = i1 & k;							// smer urejanja (padajoče: dec != 0)
		int i1s = i1 % blockDim.x;
		int i2s = i2 % blockDim.x;
		if ((dec == 0 && as[i1s] > as[i2s]) || (dec != 0 && as[i1s] < as[i2s])) {
			int temp = as[i1s];
			as[i1s] = as[i2s];
			as[i2s] = temp;
		}
	}
}

__device__ void copyToShared(int *as, int *a) {
	int i1Start = 2 * blockDim.x * blockIdx.x;
	as[threadIdx.x] = a[i1Start + threadIdx.x];	
	as[blockDim.x + threadIdx.x] = a[i1Start + blockDim.x + threadIdx.x];
}

__device__ void copyFromShared(int *a, int *as) {
	int i1Start = 2 * blockDim.x * blockIdx.x;
	a[i1Start + threadIdx.x] = as[threadIdx.x];	
	a[i1Start + blockDim.x + threadIdx.x] = as[blockDim.x + threadIdx.x];
}

extern "C" {
__global__ void bitonicSortStart(int *a, int len) {
	extern __shared__ int as[];
	copyToShared(as, a);
	for (int k = 2; k <= 2 * blockDim.x; k <<= 1) 
		for (int j = k/2; j > 0; j >>= 1) {
			bitonicSortShared(as, len, k, j);
			__syncthreads();
	}
	copyFromShared(a, as);
}

__global__ void bitonicSortMiddle(int *a, int len, int k, int j) {
	bitonicSort(a, len, k, j);
}

__global__ void bitonicSortFinish(int *a, int len, int k) {
	extern __shared__ int as[];
	copyToShared(as, a);
	for (int j = 2*blockDim.x; j > 0; j >>= 1) {
		bitonicSortShared(as, len, k, j);
		__syncthreads();
	}
	copyFromShared(a, as);
}
}